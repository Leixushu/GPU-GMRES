/*
 * This file contains the GPU based source loading and LU triangular solve
 * for transient simulation of ETBR system.
 *
 * There are some CUDA GPU kernel functions to carry out
 * the permutation job for pivoted LU factors,
 * and the parallel source interpolation on all time steps.
 *
 * Author: Xue-Xin Liu
 *         2011-Nov-16
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

//#include <cutil_inline.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>

#include "etbr.h"
#include "gpuData.h"
#include "kernels.h"

void myMemcpyD2S2(float *dst, double *src, int n)
{
  for(int i=0; i<n; i++)
    dst[i] = (float)src[i];
}

//16384

extern "C" void cudaTranSim(gpuETBR *myGPUetbr)
{
  hipEvent_t start, stop;  hipEventCreate(&start);  hipEventCreate(&stop);  float time;
  hipEventRecord(start, 0);
    
  printf("     cudaTranSim start.\n");

  int deviceCount, dev;
  hipGetDeviceCount(&deviceCount);
  hipDeviceProp_t deviceProp;
  dev=0;
  hipGetDeviceProperties(&deviceProp, dev);
  hipSetDevice(dev);
  printf("   Device %d: \"%s\" has been selected.\n", dev,deviceProp.name);

  hipblasStatus_t cublas_status=cublasInit();
  if(cublas_status != HIPBLAS_STATUS_SUCCESS)  printf("CUBLAS failed to initialize.\n");

  int numPts=myGPUetbr->numPts, q=myGPUetbr->q, m=myGPUetbr->m,// n=myGPUetbr->n,
    nIS=myGPUetbr->nIS, nVS=myGPUetbr->nVS,
    nport=myGPUetbr->nport, partLen=0, shift=0, i;
  checkCudaErrors( hipMalloc((void**)&(myGPUetbr->ipiv_dev), q*sizeof(int)) );
  checkCudaErrors( hipMalloc((void**)&(myGPUetbr->L_hCG_dev), q*q*sizeof(double)) );
  checkCudaErrors( hipMalloc((void**)&(myGPUetbr->U_hCG_dev), q*q*sizeof(double)) );
  checkCudaErrors( hipMalloc((void**)&(myGPUetbr->hC_dev), q*q*sizeof(double)) );
  double *tmpDqVecDev, *tmpDqXrDev;
  checkCudaErrors( hipMalloc((void**)&(tmpDqVecDev), q*sizeof(double)) );
  checkCudaErrors( hipMalloc((void**)&(tmpDqXrDev), q*sizeof(double)) );
  // float *tmpSqVecDev;
  // checkCudaErrors( hipMalloc((void**)&(tmpSqVecDev), q*sizeof(float)) );
  
  if(myGPUetbr->use_cuda_double) {
    if(m*(myGPUetbr->ldUt)*sizeof(double) < 400000000)
      checkCudaErrors( hipMalloc((void**)&(myGPUetbr->ut_dev), m*(myGPUetbr->ldUt)*sizeof(double)) );
    else {
      partLen = PART_LEN;//1024; //
      checkCudaErrors( hipMalloc((void**)&(myGPUetbr->ut_dev), m*partLen*sizeof(double)) );
    }
    //checkCudaErrors( hipMalloc((void**)&(myGPUetbr->V_dev), n*q*sizeof(double)) );
    checkCudaErrors( hipMalloc((void**)&(myGPUetbr->LV_dev), nport*q*sizeof(double)) );
    checkCudaErrors( hipMalloc((void**)&(myGPUetbr->Br_dev), q*m*sizeof(double)) );
    checkCudaErrors( hipMalloc((void**)&(myGPUetbr->xr_dev), q*numPts*sizeof(double)) );
    checkCudaErrors( hipMalloc((void**)&(myGPUetbr->x_dev), nport*numPts*sizeof(double)) );
    checkCudaErrors( hipMalloc((void**)&(myGPUetbr->dcVt_dev), nVS*sizeof(double)) );
  }
  if(myGPUetbr->use_cuda_single) { // SINGLE PRECISION
    if(m*(myGPUetbr->ldUt)*sizeof(float) < 400000000)
      checkCudaErrors( hipMalloc((void**)&(myGPUetbr->ut_single_dev), m*(myGPUetbr->ldUt)*sizeof(float)) );
    else {
      partLen = PART_LEN;
      checkCudaErrors( hipMalloc((void**)&(myGPUetbr->ut_single_dev), m*partLen*sizeof(float)) );
    }

    checkCudaErrors( hipMalloc((void**)&(myGPUetbr->LV_single_dev), nport*q*sizeof(float)) );
    // checkCudaErrors( hipMalloc((void**)&(myGPUetbr->L_hCG_single_dev), q*q*sizeof(float)) );
    // checkCudaErrors( hipMalloc((void**)&(myGPUetbr->U_hCG_single_dev), q*q*sizeof(float)) );
    checkCudaErrors( hipMalloc((void**)&(myGPUetbr->hC_single_dev), q*q*sizeof(float)) );
    checkCudaErrors( hipMalloc((void**)&(myGPUetbr->Br_single_dev), q*m*sizeof(float)) );
    checkCudaErrors( hipMalloc((void**)&(myGPUetbr->xr_single_dev), q*numPts*sizeof(float)) );
    checkCudaErrors( hipMalloc((void**)&(myGPUetbr->x_single_dev), nport*numPts*sizeof(float)) );
    checkCudaErrors( hipMalloc((void**)&(myGPUetbr->dcVt_single_dev), nVS*sizeof(float)) );    
  }

  hipMemcpy(myGPUetbr->ipiv_dev, myGPUetbr->ipiv_host, q*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(myGPUetbr->L_hCG_dev, myGPUetbr->L_hCG_host, q*q*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(myGPUetbr->U_hCG_dev, myGPUetbr->U_hCG_host, q*q*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(myGPUetbr->hC_dev, myGPUetbr->hC_host, q*q*sizeof(double), hipMemcpyHostToDevice);
  if(myGPUetbr->use_cuda_double) {
    //hipMemcpy(myGPUetbr->V_dev, myGPUetbr->V_host, n*q*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(myGPUetbr->LV_dev, myGPUetbr->LV_host, nport*q*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(myGPUetbr->Br_dev, myGPUetbr->Br_host, q*m*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(myGPUetbr->xr_dev, myGPUetbr->xr0_host, q*sizeof(double), hipMemcpyHostToDevice); // only ic is copied
    hipMemcpy(myGPUetbr->dcVt_dev, myGPUetbr->dcVt_host, nVS*sizeof(double), hipMemcpyHostToDevice);
  }
  if(myGPUetbr->use_cuda_single) { // SINGLE PRECISION
    hipMemcpy(myGPUetbr->LV_single_dev, myGPUetbr->LV_single_host, nport*q*sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(myGPUetbr->L_hCG_single_dev, myGPUetbr->L_hCG_single_host, q*q*sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(myGPUetbr->U_hCG_single_dev, myGPUetbr->U_hCG_single_host, q*q*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(myGPUetbr->hC_single_dev, myGPUetbr->hC_single_host, q*q*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(myGPUetbr->Br_single_dev, myGPUetbr->Br_single_host, q*m*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(myGPUetbr->xr_single_dev, myGPUetbr->xr0_single_host, q*sizeof(float), hipMemcpyHostToDevice); // only ic is copied
    hipMemcpy(myGPUetbr->dcVt_single_dev, myGPUetbr->dcVt_single_host, nVS*sizeof(float), hipMemcpyHostToDevice);
  }
  // for(int j=0; j<q; j++)  printf("  x[%d]=%6.4e\n",j,myGPUetbr->xr0_host[j]);

  /* The following section need CPU generated source info. */
  /* 
  hipMemcpy(myGPUetbr->ut_dev, myGPUetbr->ut_host, m*(numPts-1)*sizeof(double), hipMemcpyHostToDevice);
  hipblasDgemm('N', 'N', q, numPts-1, m, 1.0, myGPUetbr->Br_dev, q, myGPUetbr->ut_dev, m,
	      0.0, myGPUetbr->xr_dev+q, q); // B*u for all time steps
  */
  /********* CPU generated source info transfered. *********/

  /*******************************************************************/
  /* The following section use parallel GPU to generate source info. */
  dim3 genUtGrd(nIS, ((numPts)+BLK_SIZE_UTGEN-1)/BLK_SIZE_UTGEN);//-1
  dim3 genUtVdcGrd(nVS, ((numPts)+BLK_SIZE_UTGEN-1)/BLK_SIZE_UTGEN);//-1
  if(partLen) {
    genUtGrd.y = ((partLen)+BLK_SIZE_UTGEN-1)/BLK_SIZE_UTGEN;
    genUtVdcGrd.y = ((partLen)+BLK_SIZE_UTGEN-1)/BLK_SIZE_UTGEN;
  }
  dim3 genUtBlk(BLK_SIZE_UTGEN);
  if(partLen==0) { // *****************************
    if(myGPUetbr->use_cuda_double) {
      gen_dcVt_kernel_wrapper//<<<genUtVdcGrd, genUtBlk>>>
	(myGPUetbr->ut_dev, myGPUetbr->dcVt_dev, numPts, myGPUetbr->ldUt,//-1
         genUtVdcGrd, genUtBlk);
    }
    if(myGPUetbr->use_cuda_single) { // SINGLE PRECISION
      gen_dcVt_single_kernel_wrapper//<<<genUtVdcGrd, genUtBlk>>>
	(myGPUetbr->ut_single_dev, myGPUetbr->dcVt_single_dev, numPts, myGPUetbr->ldUt, //-1
         genUtVdcGrd, genUtBlk);
    }

    if(myGPUetbr->PWLcurExist) {
      checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PWLnumPts_dev), nIS*sizeof(int)) );
      hipMemcpy(myGPUetbr->PWLnumPts_dev, myGPUetbr->PWLnumPts_host, nIS*sizeof(int),hipMemcpyHostToDevice);
      if(myGPUetbr->use_cuda_double) {
	checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PWLtime_dev), nIS*MAX_PWL_PTS*sizeof(double)) );
	checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PWLval_dev), nIS*MAX_PWL_PTS*sizeof(double)) );
	hipMemcpy(myGPUetbr->PWLtime_dev, myGPUetbr->PWLtime_host, nIS*MAX_PWL_PTS*sizeof(double),
		   hipMemcpyHostToDevice);
	hipMemcpy(myGPUetbr->PWLval_dev, myGPUetbr->PWLval_host, nIS*MAX_PWL_PTS*sizeof(double),
		   hipMemcpyHostToDevice);
	gen_PWLut_kernel_wrapper//<<<genUtGrd, genUtBlk>>>
          (myGPUetbr->ut_dev + myGPUetbr->nVS*myGPUetbr->ldUt,
           myGPUetbr->PWLtime_dev, myGPUetbr->PWLval_dev,
           myGPUetbr->PWLnumPts_dev, myGPUetbr->tstep, numPts, myGPUetbr->ldUt,//-1
           genUtGrd, genUtBlk);
      }
      if(myGPUetbr->use_cuda_single) { // SINGLE PRECISION
	checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PWLtime_single_dev), nIS*MAX_PWL_PTS*sizeof(float)) );
	checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PWLval_single_dev), nIS*MAX_PWL_PTS*sizeof(float)) );
	hipMemcpy(myGPUetbr->PWLtime_single_dev, myGPUetbr->PWLtime_single_host,
		   nIS*MAX_PWL_PTS*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(myGPUetbr->PWLval_single_dev, myGPUetbr->PWLval_single_host,
		   nIS*MAX_PWL_PTS*sizeof(float), hipMemcpyHostToDevice);
	gen_PWLut_single_kernel_wrapper//<<<genUtGrd, genUtBlk>>>
          (myGPUetbr->ut_single_dev + myGPUetbr->nVS*myGPUetbr->ldUt,
           myGPUetbr->PWLtime_single_dev, myGPUetbr->PWLval_single_dev,
           myGPUetbr->PWLnumPts_dev, myGPUetbr->tstep, numPts, myGPUetbr->ldUt, //-1
           genUtGrd, genUtBlk);
      }
    }

    if(myGPUetbr->PULSEcurExist) {
      if(myGPUetbr->use_cuda_double) {
	checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PULSEtime_dev), nIS*5*sizeof(double)) );
	checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PULSEval_dev), nIS*2*sizeof(double)) );
	hipMemcpy(myGPUetbr->PULSEtime_dev, myGPUetbr->PULSEtime_host, nIS*5*sizeof(double),
		   hipMemcpyHostToDevice);
	hipMemcpy(myGPUetbr->PULSEval_dev, myGPUetbr->PULSEval_host, nIS*2*sizeof(double),
		   hipMemcpyHostToDevice);
	gen_PULSEut_kernel_wrapper//<<<genUtGrd, genUtBlk>>>
          (myGPUetbr->ut_dev + myGPUetbr->nVS*myGPUetbr->ldUt,
           myGPUetbr->PULSEtime_dev, myGPUetbr->PULSEval_dev,
           myGPUetbr->tstep, numPts, myGPUetbr->ldUt, //-1
           genUtGrd, genUtBlk);
      }
      if(myGPUetbr->use_cuda_single) { // SINGLE PRECISION
	checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PULSEtime_single_dev), nIS*5*sizeof(double)) );
	checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PULSEval_single_dev), nIS*2*sizeof(double)) );
	myGPUetbr->PULSEtime_single_host = (float*)malloc( nIS*5*sizeof(float));
	myGPUetbr->PULSEval_single_host = (float*)malloc( nIS*2*sizeof(float));
	myMemcpyD2S2(myGPUetbr->PULSEtime_single_host, myGPUetbr->PULSEtime_host, nIS*5);
	myMemcpyD2S2(myGPUetbr->PULSEval_single_host, myGPUetbr->PULSEval_host, nIS*2);
	hipMemcpy(myGPUetbr->PULSEtime_single_dev, myGPUetbr->PULSEtime_single_host,
		   nIS*5*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(myGPUetbr->PULSEval_single_dev, myGPUetbr->PULSEval_single_host,
		   nIS*2*sizeof(float), hipMemcpyHostToDevice);
	gen_PULSEut_single_kernel_wrapper//<<<genUtGrd, genUtBlk>>>
	  (myGPUetbr->ut_single_dev + myGPUetbr->nVS*myGPUetbr->ldUt,
	   myGPUetbr->PULSEtime_single_dev, myGPUetbr->PULSEval_single_dev,
	   myGPUetbr->tstep, numPts, myGPUetbr->ldUt, //-1
           genUtGrd, genUtBlk);
      }
    }

    if(myGPUetbr->use_cuda_double) {
      hipblasDgemm('N', 'T', q, numPts, m, 1.0, myGPUetbr->Br_dev, q,//-1
		  myGPUetbr->ut_dev, myGPUetbr->ldUt,
		  0.0, myGPUetbr->xr_dev, q); // B*u for all time steps +q
    }
    if(myGPUetbr->use_cuda_single) { // SINGLE PRECISION
      hipblasSgemm('N', 'T', q, numPts, m, 1.0, myGPUetbr->Br_single_dev, q,//-1
		  myGPUetbr->ut_single_dev, myGPUetbr->ldUt,
		  0.0, myGPUetbr->xr_single_dev, q); // B*u for all time steps +q
    }
  }
  else { // partLen > 0 ********************************************
    printf("      Due to the large number of sources and transient steps,\n");
    printf("      the parallel loading process needs to be work on\n");
    printf("      separate time segments in order to save GPU memory.\n");
    for(shift=0; shift<numPts; shift+=partLen) {//-1
      
      genUtGrd.y = partLen/BLK_SIZE_UTGEN;
      genUtVdcGrd.y = partLen/BLK_SIZE_UTGEN;
      
      
      if(myGPUetbr->use_cuda_double) {
	gen_dcVt_part_kernel_wrapper//<<<genUtVdcGrd, genUtBlk>>>
	  (myGPUetbr->ut_dev,
	   myGPUetbr->dcVt_dev, numPts, partLen, shift,//-1
           genUtVdcGrd, genUtBlk);
      }
      if(myGPUetbr->use_cuda_single) { // SINGLE PRECISION
	gen_dcVt_part_single_kernel_wrapper//<<<genUtVdcGrd, genUtBlk>>>
	  (myGPUetbr->ut_single_dev,
	   myGPUetbr->dcVt_single_dev, numPts, partLen, shift,//-1
           genUtVdcGrd, genUtBlk);
      }

      if(myGPUetbr->PWLcurExist) {
	printf("      Under Construction: part by part evaluation of PWL sources.\n"); while(!getchar()) ;
	checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PWLnumPts_dev), nIS*sizeof(int)) );
	hipMemcpy(myGPUetbr->PWLnumPts_dev, myGPUetbr->PWLnumPts_host, nIS*sizeof(int),hipMemcpyHostToDevice);
	if(myGPUetbr->use_cuda_double) {
	  checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PWLtime_dev), nIS*MAX_PWL_PTS*sizeof(double)) );
	  checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PWLval_dev), nIS*MAX_PWL_PTS*sizeof(double)) );
	  hipMemcpy(myGPUetbr->PWLtime_dev, myGPUetbr->PWLtime_host, nIS*MAX_PWL_PTS*sizeof(double),
		     hipMemcpyHostToDevice);
	  hipMemcpy(myGPUetbr->PWLval_dev, myGPUetbr->PWLval_host, nIS*MAX_PWL_PTS*sizeof(double),
		     hipMemcpyHostToDevice);
	  gen_PWLut_kernel_wrapper//<<<genUtGrd, genUtBlk>>>
            (myGPUetbr->ut_dev + myGPUetbr->nVS*myGPUetbr->ldUt,
             myGPUetbr->PWLtime_dev, myGPUetbr->PWLval_dev,
             myGPUetbr->PWLnumPts_dev,
             myGPUetbr->tstep, numPts, myGPUetbr->ldUt,//-1
             genUtGrd, genUtBlk);
	}
	if(myGPUetbr->use_cuda_single) { // SINGLE PRECISION
	  checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PWLtime_single_dev), nIS*MAX_PWL_PTS*sizeof(float)) );
	  checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PWLval_single_dev), nIS*MAX_PWL_PTS*sizeof(float)) );
	  hipMemcpy(myGPUetbr->PWLtime_single_dev, myGPUetbr->PWLtime_single_host,
		     nIS*MAX_PWL_PTS*sizeof(float), hipMemcpyHostToDevice);
	  hipMemcpy(myGPUetbr->PWLval_single_dev, myGPUetbr->PWLval_single_host,
		     nIS*MAX_PWL_PTS*sizeof(float), hipMemcpyHostToDevice);
	  gen_PWLut_single_kernel_wrapper//<<<genUtGrd, genUtBlk>>>
            (myGPUetbr->ut_single_dev + myGPUetbr->nVS*myGPUetbr->ldUt,
             myGPUetbr->PWLtime_single_dev, myGPUetbr->PWLval_single_dev,
             myGPUetbr->PWLnumPts_dev,
             myGPUetbr->tstep, numPts, myGPUetbr->ldUt,//-1
             genUtGrd, genUtBlk);
	}
      }
      if(myGPUetbr->PULSEcurExist) {
	if(myGPUetbr->use_cuda_double) {
	  checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PULSEtime_dev), nIS*5*sizeof(double)) );
	  checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PULSEval_dev), nIS*2*sizeof(double)) );
	  hipMemcpy(myGPUetbr->PULSEtime_dev, myGPUetbr->PULSEtime_host, nIS*5*sizeof(double),
		     hipMemcpyHostToDevice);
	  hipMemcpy(myGPUetbr->PULSEval_dev, myGPUetbr->PULSEval_host, nIS*2*sizeof(double),
		     hipMemcpyHostToDevice);
	  gen_PULSEut_part_kernel_wrapper//<<<genUtGrd, genUtBlk>>>
	    (myGPUetbr->ut_dev + myGPUetbr->nVS*partLen,
	     myGPUetbr->PULSEtime_dev, myGPUetbr->PULSEval_dev,
	     myGPUetbr->tstep, numPts, partLen, shift,//-1
             genUtGrd, genUtBlk);
	}
	if(myGPUetbr->use_cuda_single) { // SINGLE PRECISION
	  checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PULSEtime_single_dev), nIS*5*sizeof(double)) );
	  checkCudaErrors( hipMalloc((void**)&(myGPUetbr->PULSEval_single_dev), nIS*2*sizeof(double)) );
	  myGPUetbr->PULSEtime_single_host = (float*)malloc( nIS*5*sizeof(float));
	  myGPUetbr->PULSEval_single_host = (float*)malloc( nIS*2*sizeof(float));
	  myMemcpyD2S2(myGPUetbr->PULSEtime_single_host, myGPUetbr->PULSEtime_host, nIS*5);
	  myMemcpyD2S2(myGPUetbr->PULSEval_single_host, myGPUetbr->PULSEval_host, nIS*2);
	  hipMemcpy(myGPUetbr->PULSEtime_single_dev, myGPUetbr->PULSEtime_single_host,
		     nIS*5*sizeof(float), hipMemcpyHostToDevice);
	  hipMemcpy(myGPUetbr->PULSEval_single_dev, myGPUetbr->PULSEval_single_host,
		     nIS*2*sizeof(float), hipMemcpyHostToDevice);
	  gen_PULSEut_part_single_kernel_wrapper//<<<genUtGrd, genUtBlk>>>
	    (myGPUetbr->ut_single_dev + myGPUetbr->nVS*partLen,
	     myGPUetbr->PULSEtime_single_dev, myGPUetbr->PULSEval_single_dev,
	     myGPUetbr->tstep, numPts, partLen, shift,//-1
             genUtGrd, genUtBlk);
	}
      }

      if(shift+partLen <= numPts-1) {
	if(myGPUetbr->use_cuda_double) {
	  hipblasDgemm('N', 'T', q, partLen, m, 1.0, myGPUetbr->Br_dev, q,
		      myGPUetbr->ut_dev, partLen,
		      0.0, myGPUetbr->xr_dev+shift*q, q); // B*u for all time steps +q
	}
	if(myGPUetbr->use_cuda_single) { // SINGLE PRECISION
	  hipblasSgemm('N', 'T', q, partLen, m, 1.0, myGPUetbr->Br_single_dev, q,
		      myGPUetbr->ut_single_dev, partLen,
		      0.0, myGPUetbr->xr_single_dev+shift*q, q); // B*u for all time steps +q
	}
      }
      else {
	if(myGPUetbr->use_cuda_double) {
	  hipblasDgemm('N', 'T', q, numPts-shift, m, 1.0, myGPUetbr->Br_dev, q,//-1
		      myGPUetbr->ut_dev, partLen,
		      0.0, myGPUetbr->xr_dev+shift*q, q); // B*u for all time steps +q
	}
	if(myGPUetbr->use_cuda_single) { // SINGLE PRECISION
	  hipblasSgemm('N', 'T', q, numPts-shift, m, 1.0, myGPUetbr->Br_single_dev, q,//-1
		      myGPUetbr->ut_single_dev, partLen,
		      0.0, myGPUetbr->xr_single_dev+shift*q, q); // B*u for all time steps +q
	}
      }
    }
  }

  if(myGPUetbr->use_cuda_double) {
    hipMemcpy(myGPUetbr->xr_dev, myGPUetbr->xr0_host, q*sizeof(double), hipMemcpyHostToDevice); // only ic is copied
  }
  if(myGPUetbr->use_cuda_single) { // SINGLE PRECISION
    hipMemcpy(myGPUetbr->xr_single_dev, myGPUetbr->xr0_single_host, q*sizeof(float), hipMemcpyHostToDevice); // only ic is copied
  }

  /*************** Parallel source generation finished. ****************/
  
  if(myGPUetbr->use_cuda_double) {
    for(i=1; i<numPts; i++) {
      hipblasDgemv('N', q, q, 1.0, myGPUetbr->hC_dev, q, myGPUetbr->xr_dev+(i-1)*q, 1,
		  1.0, myGPUetbr->xr_dev+i*q, 1); // 1/h*C*x + B*u
      permute_kernel_wrapper//<<<1,q>>>
        (myGPUetbr->xr_dev+i*q, myGPUetbr->ipiv_dev, q); // pivoting
      hipblasDtrsv('L', 'N', 'U', q, myGPUetbr->L_hCG_dev, q,
		  myGPUetbr->xr_dev+i*q, 1); // L*y = b
      hipblasDtrsv('U', 'N', 'N', q, myGPUetbr->U_hCG_dev, q,
		  myGPUetbr->xr_dev+i*q, 1); // U*x = y
    }
    hipblasDgemm('N', 'N', nport, numPts, q, 1.0, myGPUetbr->LV_dev, nport, myGPUetbr->xr_dev, q,
		0.0, myGPUetbr->x_dev, nport);
    hipMemcpy(myGPUetbr->x_host, myGPUetbr->x_dev, nport*numPts*sizeof(double), hipMemcpyDeviceToHost);
  }
  if(myGPUetbr->use_cuda_single) { // SINGLE PRECISION
    myMemcpyS2Ddev_wrapper//<<<1,q>>>
      (tmpDqXrDev, myGPUetbr->xr_single_dev, q);
    for(i=1; i<numPts; i++) {
      myMemcpyS2Ddev_wrapper//<<<1,q>>>
        (tmpDqVecDev, myGPUetbr->xr_single_dev+i*q, q);
      
      hipblasDgemv('N', q, q, 1.0, myGPUetbr->hC_dev, q, tmpDqXrDev, 1,
		  1.0, tmpDqVecDev, 1); // 1/h*C*x + B*u
      permute_kernel_wrapper//<<<1,q>>>
        (tmpDqVecDev, myGPUetbr->ipiv_dev, q); // pivoting
      hipblasDtrsv('L', 'N', 'U', q, myGPUetbr->L_hCG_dev, q,
		  tmpDqVecDev, 1); // L*y = b
      hipblasDtrsv('U', 'N', 'N', q, myGPUetbr->U_hCG_dev, q,
		  tmpDqVecDev, 1); // U*x = y
      hipMemcpy(tmpDqXrDev, tmpDqVecDev, q*sizeof(double), hipMemcpyDeviceToDevice);
      myMemcpyD2Sdev_wrapper//<<<1,q>>>
        (myGPUetbr->xr_single_dev+i*q, tmpDqVecDev, q);
      /*
      hipblasSgemv('N', q, q, 1.0, myGPUetbr->hC_single_dev, q, myGPUetbr->xr_single_dev+(i-1)*q, 1,
		  1.0, myGPUetbr->xr_single_dev+i*q, 1); // 1/h*C*x + B*u
      permute_single_kernel<<<1,q>>>(myGPUetbr->xr_single_dev+i*q,
				     myGPUetbr->ipiv_dev, q); // pivoting
      hipblasStrsv('L', 'N', 'U', q, myGPUetbr->L_hCG_dev, q,
		  myGPUetbr->xr_single_dev+i*q, 1); // L*y = b
      hipblasStrsv('U', 'N', 'N', q, myGPUetbr->U_hCG_dev, q,
		  myGPUetbr->xr_single_dev+i*q, 1); // U*x = y
      */
    }
    hipblasSgemm('N', 'N', nport, numPts, q, 1.0, myGPUetbr->LV_single_dev, nport,
                myGPUetbr->xr_single_dev, q,
		0.0, myGPUetbr->x_single_dev, nport);
    hipMemcpy(myGPUetbr->x_single_host, myGPUetbr->x_single_dev, nport*numPts*sizeof(float), hipMemcpyDeviceToHost);
  }

  hipEventRecord(stop, 0); hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  printf("                GPU parallel Time: %6.4e\n", 1e-3*time); // convert from millisecond to second
  hipEventDestroy( start ); hipEventDestroy( stop );
}


extern "C" void gpuRelatedDataInit(gpuETBR *myGPUetbr)
{
  myGPUetbr->PWLnumPts_host=NULL; myGPUetbr->PWLnumPts_dev=NULL;
  myGPUetbr->PWLtime_host=NULL; myGPUetbr->PWLtime_dev=NULL;// maximum length is MAX_PWL_PTS
  myGPUetbr->PWLval_host=NULL; myGPUetbr->PWLval_dev=NULL; // maximum length is MAX_PWL_PTS

  myGPUetbr->PWLtime_single_host=NULL; myGPUetbr->PWLtime_single_dev=NULL;// maximum length is MAX_PWL_PTS
  myGPUetbr->PWLval_single_host=NULL; myGPUetbr->PWLval_single_dev=NULL; // maximum length is MAX_PWL_PTS

  myGPUetbr->PULSEtime_host=NULL; myGPUetbr->PULSEtime_dev=NULL;
  myGPUetbr->PULSEval_host=NULL; myGPUetbr->PULSEval_dev=NULL;

  myGPUetbr->PULSEtime_single_host=NULL; myGPUetbr->PULSEtime_single_dev=NULL;
  myGPUetbr->PULSEval_single_host=NULL; myGPUetbr->PULSEval_single_dev=NULL;
}

extern "C" void gpuRelatedDataFree(gpuETBR *myGPUetbr)
{
  printf("     Free memory in myGPUetbr\n");
  //int numPts=myGPUetbr->numPts, n=myGPUetbr->n, q=myGPUetbr->q, m=myGPUetbr->m;

  if(myGPUetbr->PWLnumPts_host != NULL)
    free(myGPUetbr->PWLnumPts_host);
    
  if(myGPUetbr->PWLnumPts_dev != NULL)
    hipFree(myGPUetbr->PWLnumPts_dev);

  if(myGPUetbr->PWLtime_host != NULL)
    free(myGPUetbr->PWLtime_host);

  if(myGPUetbr->PWLtime_dev != NULL)
    hipFree(myGPUetbr->PWLtime_dev);// maximum length is MAX_PWL_PTS

  if(myGPUetbr->PWLval_host != NULL)
    free(myGPUetbr->PWLval_host);
  
  if(myGPUetbr->PWLval_dev != NULL)
    hipFree(myGPUetbr->PWLval_dev); // maximum length is MAX_PWL_PTS

  if(myGPUetbr->PWLtime_single_host != NULL)
    free(myGPUetbr->PWLtime_single_host);

  if(myGPUetbr->PWLtime_single_dev != NULL)
    hipFree(myGPUetbr->PWLtime_single_dev);// maximum length is MAX_PWL_PTS

  if(myGPUetbr->PWLval_single_host != NULL)
    free(myGPUetbr->PWLval_single_host);

  if(myGPUetbr->PWLval_single_dev != NULL)
    hipFree(myGPUetbr->PWLval_single_dev); // maximum length is MAX_PWL_PTS

  if(myGPUetbr->PULSEtime_host != NULL)
    free(myGPUetbr->PULSEtime_host);

  if(myGPUetbr->PULSEtime_dev != NULL)
    hipFree(myGPUetbr->PULSEtime_dev);

  if(myGPUetbr->PULSEval_host != NULL)
    free(myGPUetbr->PULSEval_host);

  if(myGPUetbr->PULSEval_dev != NULL)
    hipFree(myGPUetbr->PULSEval_dev);

  if(myGPUetbr->PULSEtime_single_host != NULL)
    free(myGPUetbr->PULSEtime_single_host);

  if(myGPUetbr->PULSEtime_single_dev != NULL)
    hipFree(myGPUetbr->PULSEtime_single_dev);

  if(myGPUetbr->PULSEval_single_host != NULL)
    free(myGPUetbr->PULSEval_single_host);
  
  if(myGPUetbr->PULSEval_single_dev != NULL)
    hipFree(myGPUetbr->PULSEval_single_dev);

  //cublasShutdown();
  //cutilDeviceReset();
}

extern "C" void cudaTranSim_shutdown(gpuETBR *myGPUetbr)
{
  printf("     cudaTranSim shutdown\n");
  //int numPts=myGPUetbr->numPts, n=myGPUetbr->n, q=myGPUetbr->q, m=myGPUetbr->m;

  //hipFree(myGPUetbr->V_dev);
  hipFree(myGPUetbr->L_hCG_dev);
  hipFree(myGPUetbr->U_hCG_dev);
  hipFree(myGPUetbr->hC_dev);
  hipFree(myGPUetbr->Br_dev);
  hipFree(myGPUetbr->xr_dev);
  hipFree(myGPUetbr->x_dev);

  cublasShutdown();
  hipDeviceReset();
  //cutilDeviceReset();
}


  // printf("       genUtBlk(%d,%d)\n", genUtBlk.x, genUtBlk.y);
  // printf("       genUtGrd(%d,%d)\n", genUtGrd.x, genUtGrd.y);
  // printf("       genUtVdcGrd(%d,%d)\n", genUtVdcGrd.x, genUtVdcGrd.y);
  // printf("       numPts-1=%d\n",numPts-1);

    /*
    FILE *fp;  
    char filenameH[] = "testSaveUtHostETBR.bin";
    int numPts_1=numPts-1;
    fp = fopen(filenameH, "wb");
      fwrite(&m, sizeof(int), 1, fp);
      fwrite(&numPts_1, sizeof(int), 1, fp);
      fwrite(myGPUetbr->ut_host, sizeof(double), numPts_1*m, fp);
    fclose(fp);
    printf("        >>> >>> Binary data file saved in testSaveUtHostETBR.bin\n");

    hipMemcpy(myGPUetbr->ut_host, myGPUetbr->ut_dev, (myGPUetbr->ldUt)*m*sizeof(double), hipMemcpyDeviceToHost);
    char filenameD[] = "testSaveUtDevETBR.bin";
    fp = fopen(filenameD, "wb");
      fwrite(&(myGPUetbr->ldUt), sizeof(int), 1, fp);
      fwrite(&m, sizeof(int), 1, fp);
      fwrite(myGPUetbr->ut_host, sizeof(double), (myGPUetbr->ldUt)*m, fp);
    fclose(fp);
    printf("        >>> >>> Binary data file saved in testSaveUtDevETBR.bin\n");
    while(!getchar()) ;
    */
