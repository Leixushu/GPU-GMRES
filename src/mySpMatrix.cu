/*!	\file
	\brief implement the functions of the class of MySpMatrix
*/

//#include <cutil.h>
//#include <cutil_inline.h>
#include <hip/hip_runtime_api.h>

#include "SpMV.h"

void MySpMatrix::Initilize(SpMatrix &M){
	this->numRows = M.numRows;
	this->numCols = M.numCols;
	this->numNZEntries = M.numNZEntries;

	val = new float[M.numNZEntries];
	indices = new int[M.numNZEntries];
	rowIndices = new int[M.numRows + 1];

	//genCSRFormat(&M, val, rowIndices, indices);
	for (int i = 0; i < numNZEntries; i++) {
		val[i] = (M.nzentries)[i].val;
		indices[i] = (M.nzentries)[i].colNum;
	}
	for (int i = 0; i < numRows; i++) {
		rowIndices[i] = M.rowPtrs[i];
	}
	rowIndices[numRows] = M.numNZEntries;


	hipMalloc((void**)&d_val, M.numNZEntries * sizeof(float));
	hipMalloc((void**)&d_indices, M.numNZEntries * sizeof(int));
	hipMalloc((void**)&d_rowIndices, (M.numRows+1) * sizeof(int));

	hipMemcpy(d_val, val, M.numNZEntries * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_indices, indices, M.numNZEntries * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_rowIndices, rowIndices, (M.numRows+1) * sizeof(int), hipMemcpyHostToDevice);

}

// MySpMatrix::~MySpMatrix(){
// 	hipFree(d_val);
// 	hipFree(d_indices);
// 	hipFree(d_rowIndices);
// }


