/*
 * IBM Sparse Matrix-Vector Multiplication Toolkit for Graphics Processing Units
 * (c) Copyright IBM Corp. 2008, 2009.  All Rights Reserved.
 */ 
//#include <cutil.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include "config.h"
#include "SpMV.h"
#include "SpMV_inspect.h"

void allocateSparseMatrixGPU(SpMatrixGPU *spm,
		SpMatrix *m,
		float *h_val,
		int *h_rowIndices,
		int *h_indices,
		const int numRows,
		const int numCols) 
{
  //printf("Allocate Sp Matrix in GPU:\n"); // XXLiu

	int numNonZeroElements = h_rowIndices[numRows];
	int memSize_val = sizeof(float) * numNonZeroElements;
	int memSize_row = sizeof(int) * numRows;
	//int memSize_col = sizeof(float) * numCols;

#if INSPECT
	int *ins_rowIndices, *ins_indices, *h_indicesFill, *h_rowIndicesFill;
	int nnz_fill, ins_numblocks, *ins_nnzCount_block, *ins_yCount_block;
	float *h_valFill;
	int insBStat;

#if VAR_BLOCK
	insBStat = inspectVarBlock(m, &h_valFill, &h_indicesFill,
			&h_rowIndicesFill, &ins_rowIndices,
			&ins_indices, &ins_numblocks,
			&ins_nnzCount_block, &ins_yCount_block,
			&nnz_fill, (BLOCKSIZE/HALFWARP), INSPECT_BLOCK_c, VAR_COLUMN);
	if (insBStat == ERR_INSUFFICIENT_MEMORY) {
		printf("Insufficient Memory while malloc in inspectVarBlock\n"); exit(-1);
	}
#else
	insBStat = inspectBlock(m, &ins_rowIndices, &ins_indices,
			&ins_numblocks, &ins_nnzCount_block,
			&ins_yCount_block, (BLOCKSIZE/HALFWARP), INSPECT_BLOCK_c);
	if (insBStat == ERR_INSUFFICIENT_MEMORY) {
		printf("Insufficient Memory while malloc in inspectBlock\n");
		exit(-1);
	}
#endif // VAR_BLOCK

	// not inspect, will run

#endif // INSPECT

#if INSPECT_INPUT// not run
	int *ins_rowIndices, *ins_indices, *ins_inputList; 
	int ins_numblocks, ins_inputListCount;

	printf("Inspect Input Block...\n"); // XXLiu
	inspectInputBlock(m, &ins_inputList, &ins_rowIndices, &ins_indices, &ins_numblocks, &ins_inputListCount, (BLOCKSIZE/HALFWARP),HALFWARP);
#endif

	//printf("   CUDA malloc for val and indices...\n"); // XXLiu
#if (INSPECT && VAR_BLOCK && C_GLOBAL_OPT)
	checkCudaErrors(hipMalloc((void**) &(spm->d_val), sizeof(float)*nnz_fill));
	checkCudaErrors(hipMalloc((void**) &(spm->d_indices), sizeof(int)*nnz_fill));
#else// will run
	checkCudaErrors(hipMalloc((void**) &(spm->d_val), memSize_val));
	checkCudaErrors(hipMalloc((void**) &(spm->d_indices), sizeof(int)*numNonZeroElements));
#endif

	//printf("   CUDA malloc for rowIndices...\n"); // XXLiu
	checkCudaErrors(hipMalloc((void**) &(spm->d_rowIndices), memSize_row+sizeof(int)));// attention, one more element to indecate the end of the inteval

#if INSPECT
	checkCudaErrors(hipMalloc((void**) &(spm->d_ins_indices), sizeof(int)*ins_numblocks));
	checkCudaErrors(hipMalloc((void**) &(spm->d_ins_rowIndices),
                                  sizeof(int)*(1+(int)ceild(numRows,(BLOCKSIZE/HALFWARP)))));
#endif

#if INSPECT_INPUT
	hipMalloc((void**) &(spm->d_ins_indices), sizeof(int)*(ins_numblocks+1));
	hipMalloc((void**) &(spm->d_ins_rowIndices), sizeof(int)*(1+(int)ceild(numRows,(BLOCKSIZE/HALFWARP))));
	hipMalloc((void**) &(spm->d_ins_inputList), sizeof(int)*ins_inputListCount);
#endif

	//printf("   CUDA mem copy for val, indices, and rowIndices...\n"); // XXLiu
#if (INSPECT && VAR_BLOCK && C_GLOBAL_OPT)
	checkCudaErrors(hipMemcpy(spm->d_val, h_valFill, sizeof(float)*nnz_fill,
                                  hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(spm->d_indices, h_indicesFill, sizeof(int)*nnz_fill,
                                  hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(spm->d_rowIndices, h_rowIndicesFill, memSize_row+sizeof(int),
                                  hipMemcpyHostToDevice));
#else// will run
	checkCudaErrors(hipMemcpy(spm->d_val, h_val, memSize_val, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(spm->d_indices, h_indices, sizeof(int)*numNonZeroElements,
                                  hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(spm->d_rowIndices, h_rowIndices, memSize_row+sizeof(int),
                                  hipMemcpyHostToDevice));
#endif

#if INSPECT
	hipMemcpy(spm->d_ins_indices, ins_indices, sizeof(int)*ins_numblocks, hipMemcpyHostToDevice);
	hipMemcpy(spm->d_ins_rowIndices, ins_rowIndices, sizeof(int)*(1+(int)ceild(numRows,(BLOCKSIZE/HALFWARP))), hipMemcpyHostToDevice);
#endif

#if INSPECT_INPUT
	hipMemcpy(spm->d_ins_indices, ins_indices, sizeof(int)*(ins_numblocks+1), hipMemcpyHostToDevice);
	hipMemcpy(spm->d_ins_rowIndices, ins_rowIndices, sizeof(int)*(1+(int)ceild(numRows,(BLOCKSIZE/HALFWARP))), hipMemcpyHostToDevice);
	hipMemcpy(spm->d_ins_inputList, ins_inputList, sizeof(int)*ins_inputListCount, hipMemcpyHostToDevice);
#endif
}


void gpuMallocCpyCSR(SpMatrixGPU *spm,
                     float *h_val, int *h_rowIndices, int *h_indices,
                     const int numRows, const int numCols)
{
  //printf("Allocate Sp Matrix in GPU:\n"); // XXLiu

  int numNonZeroElements = h_rowIndices[numRows];
  int memSize_val = sizeof(float) * numNonZeroElements;
  int memSize_row = sizeof(int) * numRows;
  //int memSize_col = sizeof(float) * numCols;

  checkCudaErrors(hipMalloc((void**) &(spm->d_val), memSize_val));
  checkCudaErrors(hipMalloc((void**) &(spm->d_indices), sizeof(int)*numNonZeroElements));

  //printf("   CUDA malloc for rowIndices...\n"); // XXLiu
  checkCudaErrors(hipMalloc((void**) &(spm->d_rowIndices), memSize_row+sizeof(int)));// attention, one more element to indecate the end of the inteval

  checkCudaErrors(hipMemcpy(spm->d_val, h_val, memSize_val, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(spm->d_indices, h_indices, sizeof(int)*numNonZeroElements,
                            hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(spm->d_rowIndices, h_rowIndices, memSize_row+sizeof(int),
                            hipMemcpyHostToDevice));
}


void gpuMallocCpyCSRmySpM(SpMatrixGPU *spm,
                          MySpMatrix *mySpM)
{
  int numRows=mySpM->numRows, numCols=mySpM->numCols;
  int *h_rowIndices=mySpM->rowIndices;
  int *h_indices=mySpM->indices;
  float *h_val=mySpM->val;
  
  if(mySpM->isCSR)
    gpuMallocCpyCSR(spm, h_val, h_rowIndices, h_indices, numRows, numCols);
  else
    gpuMallocCpyCSR(spm, h_val, h_rowIndices, h_indices, numCols, numRows);
    
  mySpM->d_val = spm->d_val;
  mySpM->d_rowIndices = spm->d_rowIndices;
  mySpM->d_indices = spm->d_indices;
}


void freeSparseMatrixGPU(SpMatrixGPU *spm)
{
  if(spm->d_val != NULL) {
    hipFree(spm->d_val);
    spm->d_val = NULL;
  }
  if(spm->d_rowIndices != NULL) {
    hipFree(spm->d_rowIndices);
    spm->d_rowIndices = NULL;
  }
  if(spm->d_indices != NULL) {
    hipFree(spm->d_indices);
    spm->d_indices = NULL;
  }
}

void mySpMatrixFree(MySpMatrix *mySpM)
{
  if(mySpM->val != NULL) {
    free(mySpM->val);
    mySpM->val = NULL;
  }
  if(mySpM->rowIndices != NULL) {
    free(mySpM->rowIndices);
    mySpM->rowIndices = NULL;
  }
  if(mySpM->indices != NULL) {
    free(mySpM->indices);
    mySpM->indices = NULL;
  }  
}

void mySpMatrixDoubleFree(MySpMatrixDouble *mySpM)
{
  if(mySpM->val != NULL) {
    free(mySpM->val);
    mySpM->val = NULL;
  }
  if(mySpM->rowIndices != NULL) {
    free(mySpM->rowIndices);
    mySpM->rowIndices = NULL;
  }
  if(mySpM->indices != NULL) {
    free(mySpM->indices);
    mySpM->indices = NULL;
  }  
}

