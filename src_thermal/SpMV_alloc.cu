/*
 * IBM Sparse Matrix-Vector Multiplication Toolkit for Graphics Processing Units
 * (c) Copyright IBM Corp. 2008, 2009.  All Rights Reserved.
 */ 

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include "config.h"
#include "SpMV.h"
#include "SpMV_inspect.h"

void allocateSparseMatrixGPU(SpMatrixGPU *spm,
		SpMatrix *m,
		float *h_val,
		int *h_rowIndices,
		int *h_indices,
		const int numRows,
		const int numCols) 
{
	printf("Allocate Sp Matrix in GPU:\n"); // XXLiu

	int numNonZeroElements = h_rowIndices[numRows];
	int memSize_val = sizeof(float) * numNonZeroElements;
	int memSize_row = sizeof(int) * numRows;
	//int memSize_col = sizeof(float) * numCols;

#if INSPECT
	int *ins_rowIndices, *ins_indices, *h_indicesFill, *h_rowIndicesFill;
	int nnz_fill, ins_numblocks, *ins_nnzCount_block, *ins_yCount_block;
	float *h_valFill;
	int insBStat;

#if VAR_BLOCK
	insBStat = inspectVarBlock(m, &h_valFill, &h_indicesFill,
			&h_rowIndicesFill, &ins_rowIndices,
			&ins_indices, &ins_numblocks,
			&ins_nnzCount_block, &ins_yCount_block,
			&nnz_fill, (BLOCKSIZE/HALFWARP), INSPECT_BLOCK_c, VAR_COLUMN);
	if (insBStat == ERR_INSUFFICIENT_MEMORY) {
		printf("Insufficient Memory while malloc in inspectVarBlock\n"); exit(-1);
	}
#else
	insBStat = inspectBlock(m, &ins_rowIndices, &ins_indices,
			&ins_numblocks, &ins_nnzCount_block,
			&ins_yCount_block, (BLOCKSIZE/HALFWARP), INSPECT_BLOCK_c);
	if (insBStat == ERR_INSUFFICIENT_MEMORY) {
		printf("Insufficient Memory while malloc in inspectBlock\n");
		exit(-1);
	}
#endif // VAR_BLOCK

	// not inspect, will run

#endif // INSPECT

#if INSPECT_INPUT// not run
	int *ins_rowIndices, *ins_indices, *ins_inputList; 
	int ins_numblocks, ins_inputListCount;

	printf("Inspect Input Block...\n"); // XXLiu
	inspectInputBlock(m, &ins_inputList, &ins_rowIndices, &ins_indices, &ins_numblocks, &ins_inputListCount, (BLOCKSIZE/HALFWARP),HALFWARP);
#endif

	printf("   CUDA malloc for val and indices...\n"); // XXLiu
#if (INSPECT && VAR_BLOCK && C_GLOBAL_OPT)
	hipMalloc((void**) &(spm->d_val), sizeof(float)*nnz_fill);
	hipMalloc((void**) &(spm->d_indices), sizeof(int)*nnz_fill);
#else// will run
	hipMalloc((void**) &(spm->d_val), memSize_val);
	hipMalloc((void**) &(spm->d_indices), sizeof(int)*numNonZeroElements);
#endif

	printf("   CUDA malloc for rowIndices...\n"); // XXLiu
	hipMalloc((void**) &(spm->d_rowIndices), memSize_row+sizeof(int));// attention, one more element to indecate the end of the inteval

#if INSPECT
	hipMalloc((void**) &(spm->d_ins_indices), sizeof(int)*ins_numblocks);
	hipMalloc((void**) &(spm->d_ins_rowIndices), sizeof(int)*(1+(int)ceild(numRows,(BLOCKSIZE/HALFWARP))));
#endif

#if INSPECT_INPUT
	hipMalloc((void**) &(spm->d_ins_indices), sizeof(int)*(ins_numblocks+1));
	hipMalloc((void**) &(spm->d_ins_rowIndices), sizeof(int)*(1+(int)ceild(numRows,(BLOCKSIZE/HALFWARP))));
	hipMalloc((void**) &(spm->d_ins_inputList), sizeof(int)*ins_inputListCount);
#endif

	printf("   CUDA mem copy for val, indices, and rowIndices...\n"); // XXLiu
#if (INSPECT && VAR_BLOCK && C_GLOBAL_OPT)
	hipMemcpy(spm->d_val, h_valFill, sizeof(float)*nnz_fill, hipMemcpyHostToDevice);
	hipMemcpy(spm->d_indices, h_indicesFill, sizeof(int)*nnz_fill, hipMemcpyHostToDevice);
	hipMemcpy(spm->d_rowIndices, h_rowIndicesFill, memSize_row+sizeof(int), hipMemcpyHostToDevice);
#else// will run
	hipMemcpy(spm->d_val, h_val, memSize_val, hipMemcpyHostToDevice);
	hipMemcpy(spm->d_indices, h_indices, sizeof(int)*numNonZeroElements, hipMemcpyHostToDevice);
	hipMemcpy(spm->d_rowIndices, h_rowIndices, memSize_row+sizeof(int), hipMemcpyHostToDevice);
#endif

#if INSPECT
	hipMemcpy(spm->d_ins_indices, ins_indices, sizeof(int)*ins_numblocks, hipMemcpyHostToDevice);
	hipMemcpy(spm->d_ins_rowIndices, ins_rowIndices, sizeof(int)*(1+(int)ceild(numRows,(BLOCKSIZE/HALFWARP))), hipMemcpyHostToDevice);
#endif

#if INSPECT_INPUT
	hipMemcpy(spm->d_ins_indices, ins_indices, sizeof(int)*(ins_numblocks+1), hipMemcpyHostToDevice);
	hipMemcpy(spm->d_ins_rowIndices, ins_rowIndices, sizeof(int)*(1+(int)ceild(numRows,(BLOCKSIZE/HALFWARP))), hipMemcpyHostToDevice);
	hipMemcpy(spm->d_ins_inputList, ins_inputList, sizeof(int)*ins_inputListCount, hipMemcpyHostToDevice);
#endif
}

